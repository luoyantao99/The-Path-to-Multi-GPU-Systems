#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

__global__ void MaxSum(float *,float *,float *,float *,int,int,int);
__global__ void SecondReduc(float *,float *);

#define index(i, j, k)  ((i)*(S)*(A)) + ((j)*(S)) + (k)

int main(int argc, char * argv[])
{
  int threadperblock = 64;
  float * FullT;
  float * FullR;
  float * StateT;
  float * StateR;
  float * BlockMaxs;
  float * BlockMaxsCPU;
  float * StateMax;
  float * StateMaxCPU;
  float * next;
  float * V;
  float * T;
  float * R;
  int Sint;
  int Aint;

  unsigned int S;
  unsigned int A;
  int numiters = 0;

  numiters = atoi(argv[3]);
  A =  (unsigned int) atoi(argv[1]);
  S =  (unsigned int) atoi(argv[2]);

  /**** Fill T and R ******/
  FullT = (float *)calloc(S*A*S, sizeof(float));
  FullR = (float *)calloc(S*A*S, sizeof(float));

  for (int tr = 0; tr<S*A*S;tr++){
    FullT[tr] = 1;
    FullR[tr] = 1;
  }


  /**** Fill V ******/
  next = (float *)calloc(S, sizeof(float));
  for (int j = 0; j<S;j++){
    next[j] = 0;
  }

  StateT = (float *)calloc(S*A, sizeof(float));
  StateR = (float *)calloc(S*A, sizeof(float));

  /*** Allocate Required Space on GPU ***/
  hipMalloc((void **)&V, S*sizeof(float));
  hipMalloc((void **)&T, S*A*sizeof(float));
  hipMalloc((void **)&R, S*A*sizeof(float));
  hipMalloc((void **)&BlockMaxs, ceil(A/threadperblock)*sizeof(float));
  hipMalloc((void **)&StateMax, 1*sizeof(float));

  /**** Loop Through Iterations ******/
  for (int i = 0; i < numiters; i++) {
    printf("i = %u\n", i);
  /*** Move next to V ***/
  hipMemcpy(V, next, S*sizeof(float), hipMemcpyHostToDevice);

  /*** Loop Through States per GPU ***/
  for (int k = 0; k < S; k++) {


    /*** Find the part of the full T and R arrays that are needed for this state. Fill new vectors ***/
    for (int j = 0; j < S*A; j++) {
      StateT[j] = FullT[k*Sint*Aint + j];
      StateR[j] = FullR[k*S*A + j];
    }

    /*** Move all required arrays to GPU ***/
    hipMemcpy(T, StateT, S*A*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(R, StateR, S*A*sizeof(float), hipMemcpyHostToDevice);

    /*** Run the Max and Sum for each block at a time. The number of total thread = number of actions.  ***/
    MaxSum<<<ceil(A/threadperblock), threadperblock>>>(BlockMaxs,V,R,T,k,A,S);
    hipDeviceSynchronize();
    /*** Use second kernel to find max of all blocks.  ***/
    SecondReduc<<<1, ceil(A/threadperblock)/2>>>(StateMax,BlockMaxs);
    hipDeviceSynchronize();

    /*** Save the state max. ***/
    hipMemcpy(StateMaxCPU, StateMax, 1*sizeof(float), hipMemcpyDeviceToHost);
    next[k] = StateMaxCPU[0];
  }
  // Synchronize all the GPUs before beginning next iteration.
  hipDeviceSynchronize();
  }

}

__global__  void MaxSum(float *BlockMaxs,float * V, float * R,float * T,int sID,int A,int S)
{
    // Use Shared Memory to write sums
    __shared__ float sprimeSumValues[64];
    float sprimeSum;
    int aID;
    sprimeSum = 0;
    // Action ID
    aID = blockIdx.x*blockDim.x + threadIdx.x;

    // Loop all s' and perform sum
    for (int spID = 0; spID < S; spID ++)
    {
      sprimeSum += T[aID*S + spID]*(R[aID*S + spID] + V[spID]);
    }
    // Save s prime sum value
    sprimeSumValues[threadIdx.x] = sprimeSum;

    //Wait till all threads have done this
    __syncthreads();
    // Use Reduction Tree to quickly find max of each block
    for ( int stride = blockDim.x/2; stride >= 1; stride /= 2)
    {
      __syncthreads();
      if (threadIdx.x < stride) {            // aID is thread (action) ID
       sprimeSumValues[threadIdx.x] = max(sprimeSumValues[threadIdx.x],sprimeSumValues[threadIdx.x + stride]);
    }
    }
    if (threadIdx.x == 0 ){
      // Save the block max and then find total max somehow
      BlockMaxs[blockIdx.x] = sprimeSumValues[0];
    }
 }

 __global__  void SecondReduc(float * StateMax, float *BlockMaxs)
 {
     __syncthreads();
     // Use Reduction Tree to quickly find max of all blocks
     for (int stride = blockDim.x; stride >= 1; stride /= 2)
     {
       __syncthreads();
       if (threadIdx.x < stride) {
        BlockMaxs[threadIdx.x] = max(BlockMaxs[threadIdx.x],BlockMaxs[threadIdx.x + stride]);
     }
     }
     // Save the state max
     if (threadIdx.x == 0 ) {
      StateMax[0] = BlockMaxs[0];
    }
  }
