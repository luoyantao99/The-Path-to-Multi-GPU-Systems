#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

__global__ void MaxSum(float *,float *,float *,float *,int,int,int);
__global__ void SecondReduc(float *,float *);

int main(int argc, char * argv[])
{
  int threadperblock = 64;
  float * FullT;
  float * FullR;
  float * BlockMaxArrays;
  float * V;
  int numDevs= 0;

  unsigned int S;
  unsigned int A;
  int numiters = 0;

  numiters = atoi(argv[3]);
  A =  (unsigned int) atoi(argv[1]);
  S =  (unsigned int) atoi(argv[2]);

  /**** Fill T and R ******/
  // Allocate Unified Memory
  hipMallocManaged(&FullT, S*A*S*sizeof(float));
  hipMallocManaged(&FullR, S*A*S*sizeof(float));
  hipMallocManaged(&V, S*sizeof(float));
  hipMallocManaged(&BlockMaxArrays, S*ceil(A/threadperblock)*sizeof(float));

  printf("i = \n");

  for (int tr = 0; tr<S*A*S;tr++){
    FullT[tr] = 1;
    FullR[tr] = 1;
  }
  printf("i = \n");

  /**** Fill next ******/
  for (int j = 0; j<S;j++){
    V[j] = 0;
  }
  printf("i = \n");

  // Find number of GPUs
  hipGetDeviceCount(&numDevs);

  /**** Loop Through Iterations ******/
  for (int i = 0; i < numiters; i++) {
    printf("i = %u\n", i);

  /*** Loop Through States per GPU ***/
  for (int k = 0; k < S/numDevs; k++) {

  /*** Loop Through GPU Devices ***/
  for (int d = 0; d < numDevs; d++) {

    hipSetDevice(d);

    /*** Run the Max and Sum for each block at a time. The number of total thread = number of actions.  ***/
    MaxSum<<<ceil(A/threadperblock), threadperblock>>>(BlockMaxArrays,V,FullR,FullT,k*numDevs+d,A,S);
  }
  }

  for (int d = 0; d < numDevs; d++) {
    hipDeviceSynchronize();
  }

  /*** Use second kernel to find max of all blocks.  ***/
  SecondReduc<<< S , ceil(A/threadperblock)/2>>>(V,BlockMaxArrays);
  hipDeviceSynchronize();
  }

}

__global__  void MaxSum(float *BlockMaxArrays,float * V, float * FullR,float * FullT,int sID,int A,int S)
{
    // Use Shared Memory to write sums
    __shared__ float sprimeSumValues[64];
    float sprimeSum;
    int aID;
    sprimeSum = 0;
    // Action ID
    aID = blockIdx.x*blockDim.x + threadIdx.x;

    // Loop all s' and perform sum
    for (int spID = 0; spID < S; spID ++)
    {
      sprimeSum += FullT[S*A*sID + aID*S + spID]*(FullR[S*A*sID + aID*S + spID] + V[spID]);
    }
    // Save s prime sum value
    sprimeSumValues[threadIdx.x] = sprimeSum;

    //Wait till all threads have done this
    __syncthreads();
    // Use Reduction Tree to quickly find max of each block
    for ( int stride = blockDim.x/2; stride >= 1; stride /= 2)
    {
      __syncthreads();
      if (threadIdx.x < stride) {            // aID is thread (action) ID
       sprimeSumValues[threadIdx.x] = max(sprimeSumValues[threadIdx.x],sprimeSumValues[threadIdx.x + stride]);
    }
    }
    if (threadIdx.x == 0 ){
      // Save the block max and then find total max somehow
      BlockMaxArrays[sID*gridDim.x + blockIdx.x] = sprimeSumValues[0];
    }
 }

 __global__  void SecondReduc(float * V, float *BlockMaxArrays)
 {
     __syncthreads();
     // Use Reduction Tree to quickly find max of all blocks
     for (int stride = blockDim.x; stride >= 1; stride /= 2)
     {
       __syncthreads();
       if (threadIdx.x < stride) {
         BlockMaxArrays[blockIdx.x*blockDim.x*2 + threadIdx.x] = max(BlockMaxArrays[blockIdx.x*blockDim.x*2 + threadIdx.x],BlockMaxArrays[blockIdx.x*blockDim.x*2 + threadIdx.x + stride]);
     }
     }
     // Save the state max
     if (threadIdx.x == 0) {
      V[blockIdx.x] = BlockMaxArrays[blockIdx.x*blockDim.x*2];
    }
  }
